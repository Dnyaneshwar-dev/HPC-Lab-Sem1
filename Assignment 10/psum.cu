#include "hip/hip_runtime.h"
#include<stdio.h>


__global__ void scan(float *g_odata, float *g_idata, int n)
{
 extern __shared__ float temp[]; // allocated on invocation
 int thid = threadIdx.x;
 int pout = 0, pin = 1;
 // load input into shared memory.
 // This is exclusive scan, so shift right by one and set first elt to 0
 temp[pout*n + thid] = (thid > 0) ? g_idata[thid-1] : 0;
 __syncthreads();
 for (int offset = 1; offset < n; offset *= 2)
 {
 pout = 1 - pout; // swap double buffer indices
 pin = 1 - pout;
 if (thid >= offset)
 temp[pout*n+thid] += temp[pin*n+thid - offset];
 else
 temp[pout*n+thid] = temp[pin*n+thid];
 __syncthreads();
 }
 g_odata[thid] = temp[pout*n+thid1]; // write output
} 



void initWith(float val, float *arr, int N)
{
  for (int i = 0; i < N; i++)
  {
    arr[i] = val;
  }
}

void checkRes(float *arr, float *res, int N, float *ptemp, float* ttemp)
{
  float sum = 0;
  for (int i = 0; i < N; i++)
  {
    sum += arr[i];
    if (sum != res[i])
    {
      printf("FAIL: res[%d] - %0.0f does not equal %0.0f\n", i, res[i], sum);
      exit(1);
    }
  }
  printf("SUCCESS! All prefix sums added correctly.\n");
}

int main()
{
  const int N = 1000000;
  size_t size = N * sizeof(float);

  float *arr;
  float *res;

  hipMallocManaged(&arr, size);
  hipMallocManaged(&res, size);

  initWith(2, arr, N);
  initWith(0, res, N);

  int blocks = 1;
  int threadsPerBlock = 32;
  int totalThreads = blocks * threadsPerBlock;

  float *ptemp;
  float *ttemp;
  hipMallocManaged(&ptemp, totalThreads * sizeof(float));
  hipMallocManaged(&ttemp, totalThreads * sizeof(float));

  scan<<<blocks, threadsPerBlock>>>(res,arr,N);
  hipDeviceSynchronize();

  checkRes(arr, res, N, ptemp, ttemp);

  hipFree(arr);
  hipFree(res);
  hipFree(ttemp);
  hipFree(ptemp);
}